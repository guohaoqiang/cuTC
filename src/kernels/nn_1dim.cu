#include "hip/hip_runtime.h"
__device__
void tc_1dim_64_8X8_NN(float* A, int* dim_A, int* inter_A, int* acc_dims_A, int nd_A, \
                    float* B, int* dim_B, int* inter_B, int* acc_dims_B, int nd_B, \
                    float* C, int* dim_C_A, int* acc_dims_C_A, int nd_C_A, int* dim_C_B, int* acc_dims_C_B, int nd_C_B, int Mask, \
                    int64_t* dim, int nd){
    
    // A: {dim[dim_A[0]], dim[dim_A[1]], dim[dim_A[2]], ...}
    // B: {dim[dim_B[0]], dim[dim_B[1]], dim[dim_B[2]], ...}
    // C: {dim[dim_C[0]], dim[dim_C[1]], dim[dim_C[2]], ...}
    int BY = 4, BX = 4, BK = 8;
    extern __shared__ float sh[]; 
    // shared_mem: BY*BK + BX*BK | BY*BK + BX*BK
    float *sh_A = sh;
	float *sh_B = sh + (BY*BK);  // maybe needs to modified
//------------------------------C---------------------------    
    // C (a,b  X  c,d)
    // horizontal
    int blocks_x = (dim[dim_C_B[0]] + BX - 1)/BX;
    
    // vertical
	int blocks_y = (dim[dim_C_A[0]] + BY - 1)/BY;
 
    // Load C from global memory to register file
    // Here, blockDim.x = BX * BY
    //                   + a                                               + b * a_dims                     + c * a_dims * b_dims                                               + d * a_dims * b_dims * c_dims
	//float *C_start = C + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * a_dims * b_dims + (blockIdx.x / blocks_x) * a_dims * b_dims * c_dims ;
	int idx[8]; // len is nd, which is the same with dim array
    // BY <= dim[dim_C_A[0]]
    int coeff_y = blockIdx.y % blocks_y * BY + threadIdx.x % BY; 
    // BX <= dim[dim_C_B[0]]
    int coeff_x = blockIdx.x % blocks_x * BX + threadIdx.x / BY;
    
    float reg_C; 
    int C_offset = 0;
    if ( coeff_y < dim[dim_C_A[0]] && coeff_x < dim[dim_C_B[0]] ){
        idx[dim_C_A[0]] = coeff_y;
        idx[dim_C_B[0]] = coeff_x;

        C_offset += idx[dim_C_A[0]];
        for (int i=1; i<nd_C_A; ++i){
            idx[dim_C_A[i]] = blockIdx.y / (blocks_y * acc_dims_C_A[i-1]);
            C_offset += idx[dim_C_A[i]] * acc_dims_C_A[i];
        }

        C_offset += idx[dim_C_B[0]] * acc_dims_C_B[0] * acc_dims_C_A[nd_C_A];
        for (int i=1; i<nd_C_B; ++i){
            idx[dim_C_B[i]] = blockIdx.x / (blocks_x * acc_dims_C_B[i-1]);
            C_offset += idx[dim_C_B[i]] * acc_dims_C_B[i] * acc_dims_C_A[nd_C_A];
        }
        //C_start = C;
        reg_C = C[C_offset]; 
    }    
//----------------------------------------------------------    
//------------------------------A---------------------------    
    // A (a e b f)
    //load A from global memory to shared memory
    //                   + a                                               + b * a_dims * e_dims                     + (e) 0 * a_dims  + (f) 0 * a_dims * e_dims * b_dims
    //var/folders/6_/_434ff9j2cz1f0psvgwdf9700000gn/T/TemporaryItems/NSIRD_screencaptureui_XG5uhF/Screen\ Shot\ 2022-01-06\ at\ 2.46.30\ PM.png /float *A_start = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims + 0 * a_dims      + threadIdx.x / BY * a_dims * e_dims * b_dims;
    //int A_base = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims;   // external indices
    //float *A_start = A_base + 0 * a_dims      + (threadIdx.x / BY+0) * a_dims * e_dims * b_dims;    // internal indices of A
    int A_internal_offset = 0;
    int A_base = 0;
    
    // the first dim of A is an external dim. A1 case
    for (int i=0; i<nd_A-1; ++i){
        // the dim belongs to external dims
        A_base += idx[dim_A[i]]*acc_dims_A[i];
    }
    // the dim belongs to internal dims
    int d = threadIdx.x / BY;
    if (d < BK){
        // <(less than) internal dims
        A_internal_offset += d * acc_dims_A[nd_A-1];
    }
    A_internal_offset += A_base;
    if ( threadIdx.x < BY*BK ){
        *(sh_A + threadIdx.x) = A[A_internal_offset];
//----------------------------------------------------------    
//------------------------------B---------------------------    
    // B (f c e d)
    //load B from global memory to shared memory
    //                 + c * f_dims                                               +  d * f_dims * c_dims *e_dims                      + (e) 0 * f_dims * c_dims    + (f) 0
    //float *B_start = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims + 0 * f_dims * c_dims        + threadIdx.x % BK;
    //int B_base = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims;   // external indices of B
    //float *B_start = B_base + 0 * f_dims * c_dims        + threadIdx.x % BK * 1;     // internal indices of B
    
    int B_internal_offset = 0;
    int B_base = 0;
    // the first dim of B is a contraction dim. B1 case
    // in this case, we have select it as the first iteration contraction dim previously
    if (threadIdx.x / BK < BX){
        B_internal_offset += threadIdx.x % BK;
        for (int i=1; i<nd_B; ++i){
            // all the remaining dims are external dims
            B_base += idx[dim_B[i]]*acc_dims_B[i];
        }
    }
    B_internal_offset += B_base;
    if (threadIdx.x < BX*BK){
        *(sh_B + threadIdx.x) = B[B_internal_offset];
    }
//----------------------------------------------------------    
}
