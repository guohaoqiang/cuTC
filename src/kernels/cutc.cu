#include "hip/hip_runtime.h"
#include <hiptensor.h>
#include <hip/hip_runtime.h>
#include "../../include/cutc.cuh"
#include "../../include/util.cuh"
// A[], (0,4,1,5), (1,3), (1, a_dims, a_dims*e_dims, a_dims*e_dims*b_dims), nd_A = 4
// B[], (5,2,4,3), (0,2), (1, f_dims, f_dims*c_dims, f_dims*c_dims*e_dims), nd_B = 4
// (a_dims,b_dims,c_dims,d_dims,e_dims,f_dims), nd = 6
// C[], (0,1) (2,3), (001111), (1, a_dims, a_dims*b_dims, a_dims*b_dims*c_dims), nd_C = 4
// acc_dims_C_A: (1,a_dims,a_dims*b_dims)   len is nd_C_A + 1
// acc_dims_C_B: (1,c_dims,c_dims*d_dims)   len is nd_C_B + 1
__global__ 
void tc_2dims_64_8X8(float A[], int dim_A[], int inter_A[], int acc_dims_A[], int nd_A, \
                    float B[], int dim_B[], int inter_B[], int acc_dims_B[], int nd_B, \
                    float C[], int dim_C_A[], int acc_dims_C_A[], int nd_C_A, int dim_C_B[], int acc_dims_C_B[], int nd_C_B, int Mask, \
                    int64_t dim[], int nd){
    // A: {dim[dim_A[0]], dim[dim_A[1]], dim[dim_A[2]], ...}
    // B: {dim[dim_B[0]], dim[dim_B[1]], dim[dim_B[2]], ...}
    // C: {dim[dim_C[0]], dim[dim_C[1]], dim[dim_C[2]], ...}
    int BY = 8, BX = 8, BK = 8;
    extern __shared__ float sh[]; 
    // shared_mem: BY*BK + BX*BK | BY*BK + BX*BK
    float *sh_A = sh;
	float *sh_B = sh + (BY*BK);  // maybe needs to modified
    
    // C (a,b  X  c,d)
    // horizontal
    //int blocks_x = (c_dims + BX - 1)/BX;
    int blocks_x = (dim[dim_C_B[0]] + BX - 1)/BX;
    
    // vertical
    //int blocks_y = (a_dims + BY - 1)/BY;
	int blocks_y = (dim[dim_C_A[0]] + BY - 1)/BY;
    
    // Load C from global memory to register file
    // Here, blockDim.x = BX * BY
    //                   + a                                               + b * a_dims                     + c * a_dims * b_dims                                               + d * a_dims * b_dims * c_dims
	//float *C_start = C + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * a_dims * b_dims + (blockIdx.x / blocks_x) * a_dims * b_dims * c_dims ;
	int idx[8]; // len is nd, which is the same with dim array
    idx[dim_C_A[0]] = (blockIdx.y % blocks_y * BY + threadIdx.x % BY);
    float *C_start = C + idx[dim_C_A[0]];
    for (int i=1; i<nd_C_A; ++i){
        idx[dim_C_A[i]] = blockIdx.y / (blocks_y * acc_dims_C_A[i-1]);
        C_start += idx[dim_C_A[i]] * acc_dims_C_A[i];
    }
    idx[dim_C_B[0]] = (blockIdx.x % blocks_x * BX + threadIdx.x / BY);
    C_start += idx[dim_C_B[0]] * acc_dims_C_B[0] * acc_dims_C_A[nd_C_A];
    for (int i=1; i<nd_C_B; ++i){
        idx[dim_C_B[i]] = blockIdx.x / (blocks_x * acc_dims_C_B[i-1]);
        C_start += idx[dim_C_B[i]] * acc_dims_C_B[i] * acc_dims_C_A[nd_C_A];
    }
    float reg_C = *C_start; 
    
    // A (a e b f)
    //load A from global memory to shared memory
    //                   + a                                               + b * a_dims * e_dims                     + (e) 0 * a_dims  + (f) 0 * a_dims * e_dims * b_dims
    //float *A_start = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims + 0 * a_dims      + threadIdx.x / BY * a_dims * e_dims * b_dims;
    //int A_base = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims;   // external indices
    //float *A_start = A_base + 0 * a_dims      + (threadIdx.x / BY+0) * a_dims * e_dims * b_dims;    // internal indices of A
    float *A_start = A;
    int first_internal_dim = -1;
    int A_base = 0;
    if (dim_A[0]==dim_C_A[0]){
        // the first dim of A is an external dim. A1 case
        A_start += idx[dim_A[0]];
        for (int i=1; i<nd_A; ++i){
            // the dim belongs to external dims
            if ( (1<<dim_A[i]) & Mask ) A_base += idx[dim_A[i]]*acc_dims_A[i];
            // the dim belongs to internal dims
            else {
            // the first dimension of B is a contraction dim (f dim of B1 in our case), then we iterate the contraction dim of A first. B1 case
            // A1 * B1
                if (dim_B[0]!=dim_C_B[0]){
                    // f dim of A1
                    if (dim_A[i]==dim_C_B[0]){
                        A_start += threadIdx.x / (BY) * acc_dims_A[i];
                        first_internal_dim = dim_A[i];
                    }
                    // e dim of A1
                    else{
                        A_start += 0 * acc_dims_A[i];
                    }
                }
            // the first dimension of B is not a contraction dim, then we iterate the first contraction dim of A first (e dim of A1 in our case). B2 case
            // A1 * B2
                else{
                    // e dim of A
                    if (first_internal_dim == -1){
                        first_internal_dim = dim_A[i];
                        A_start += threadIdx.x / (BY) * acc_dims_A[i];
                    }
                    // f dim of A
                    else{
                        A_start += 0 * acc_dims_A[i];
                    }
                }
            }  
        }
    }else{
        // the first dim of A is a contraction dim. A2 case
        A_start += threadIdx.x % BK;
        // the first contradiction dim of A2. e dim of A2
        first_internal_dim = dim_A[0]; 
        for (int i=1; i<nd_A; ++i){
            // the dim belongs to external dims
            if ( (1<<dim_A[i]) & Mask ) A_base += idx[dim_A[i]] * acc_dims_A[i];
            // the dim belongs to internal dims
            else {
                // the second contraction dim. f dim of A2
                A_start += 0*acc_dims_A[i];
            }
        }
    }
    A_start += A_base;
    *(sh_A + threadIdx.x) = *(A_start);
    // B (f c e d)
    //load B from global memory to shared memory
    //                 + c * f_dims                                               +  d * f_dims * c_dims *e_dims                      + (e) 0 * f_dims * c_dims    + (f) 0
    //float *B_start = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims + 0 * f_dims * c_dims        + threadIdx.x % BK;
    //int B_base = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims;   // external indices of B
    //float *B_start = B_base + 0 * f_dims * c_dims        + threadIdx.x % BK * 1;     // internal indices of B
    
    float *B_start = B;
    int B_base = 0;
    if (dim_B[0]!=dim_C_B[0]){
        // the first dim of B is a contraction dim. B1 case
        // in this case, we have select it as the first iteration contraction dim previously
        B_start += threadIdx.x % BK;
        for (int i=1; i<nd_B; ++i){
            // the dim belongs to external dims
            if ( (1<<dim_B[i]) & Mask ) B_base += idx[dim_B[i]]*acc_dims_B[i];
            else{
                B_start += 0;
            } 
        }
    }else{
        // the first dim of B is an external dim. B2 case
        B_start += idx[dim_B[0]]*acc_dims_B[0];
        for (int i=1; i<nd_B; ++i){
            // the dim belongs to external dims
            if ( (1<<dim_B[i]) & Mask ) B_base += idx[dim_B[i]]*acc_dims_B[i];
            else{
                if (dim_B[i]==first_internal_dim){
                    B_start += threadIdx.x / BK;
                }else{
                    B_start += 0*acc_dims_B[i];
                }
            } 
        }

    }
    B_start += B_base;
    *(sh_B + threadIdx.x) = *(B_start);
    
    // compute the number of contraction dims
    int prod = dim[dim_A[inter_A[0]]] * dim[dim_A[inter_A[1]]];
    
    // shared_mem: (BY*BK + BX*BK) * 2
    //        0    ~ (BY-1)*BK: A
    // (BY-1)*BK+1 ~ (BY-1)*BK+(BX-1)*BK: B
    int double_buffer = 0; 
    float reg_A, reg_B;
    for (int i=0; i<prod; i += BK){
        __syncthreads();
        int A_offset = double_buffer + (threadIdx.x%BY);
		int B_offset = double_buffer + (threadIdx.x/BY)*BK;
            
        for (int k=0; k<BK; ++k){
            // read A tile from shared memory to registers
            reg_A = sh_A[A_offset];
            // read B tile from shared memory to registers
            reg_B = sh_B[B_offset];
            // perform FMA
            reg_C = fma(reg_A, reg_B, reg_C);

            A_offset += BY;
			B_offset += 1;
        }
        // BX*BK+BY*BK = 8*8+8*8 = 128
        double_buffer ^= (BX*BK+BY*BK);
        if (i+BK < prod){
            if (dim_C_A[0]==dim_A[0]){
                /*
                if (first_internal_dim==dim_A[inter_A[0]]){
                    // internal indices of A. A1 case
                    //f_A = (threadIdx.x/BY + i + BK) % first_internal_dim;
                    //e_A = (threadIdx.x/BY + i + BK) / first_internal_dim;
                    //               + e * a_dims + f * a_dims * e_dims * b_dims
                    A_start = A_base + (threadIdx.x/BY + i + BK) % dim[first_internal_dim] * acc_dims_A[inter_A[0]] + 
                                (threadIdx.x/BY + i + BK) / dim[first_internal_dim] * acc_dims_A[inter_A[1]]; 
                }else{
                    A_start = A_base + (threadIdx.x/BY + i + BK) % dim[first_internal_dim] * acc_dims_A[inter_A[1]] + 
                                (threadIdx.x/BY + i + BK) / dim[first_internal_dim] * acc_dims_A[inter_A[0]];
                }*/
                // the above if..else.. can be combined as one
                A_start = A + A_base + (threadIdx.x/BY + i + BK) % dim[dim_A[inter_A[1]]] * acc_dims_A[inter_A[1]] + 
                                (threadIdx.x/BY + i + BK) / dim[dim_A[inter_A[0]]] * acc_dims_A[inter_A[0]];
            }else{
                // A2 case
                //f_A = (threadIdx.x%BK + i + BK) % first_internal_dim;
                //e_A = (threadIdx.x%BK + i + BK) / first_internal_dim; 
                A_start = A + A_base + (threadIdx.x/BK + i + BK) % dim[first_internal_dim] * acc_dims_A[inter_A[0]] + 
                            (threadIdx.x/BK + i + BK) / dim[first_internal_dim] * acc_dims_A[inter_A[1]]; 
            }
            *(sh_A + double_buffer + threadIdx.x) = *(A_start);

            if (dim_B[0]==first_internal_dim){
                // internal indices of B. B1 case
                //f_B = (threadIdx.x%BK + i + BK) % first_internal_dim;
                //e_B = (threadIdx.x%BK + i + BK) / first_internal_dim;
                //               + e * f_dims * c_dims + f
                B_start = B + B_base + (threadIdx.x/BK + i + BK) % dim[first_internal_dim] * acc_dims_B[inter_B[0]] + 
                (threadIdx.x/BK + i + BK) / dim[first_internal_dim] * acc_dims_B[inter_B[1]];  
            }else{
                /*
                if (first_internal_dim==dim_B[inter_B[0]]){
                    //f_A = (threadIdx.x/BX + i + BK) % first_internal_dim;
                    //e_A = (threadIdx.x/BX + i + BK) / first_internal_dim;
                    //               + e * f_dims * c_dims + f
                    B_start = B_base + (threadIdx.x/BX + i + BK) % dim[first_internal_dim] * acc_dims_B[inter_B[0]] + 
                                (threadIdx.x/BX + i + BK) / dim[first_internal_dim] * acc_dims_B[inter_B[1]];
                }else{
                    B_start = B_base + (threadIdx.x/BX + i + BK) % dim[first_internal_dim] * acc_dims_B[inter_B[1]] + 
                                (threadIdx.x/BX + i + BK) / dim[first_internal_dim] * acc_dims_B[inter_B[0]];
                }*/
                // the above if..else.. can be combined as one
                B_start = B + B_base + (threadIdx.x/BX + i + BK) % dim[dim_B[inter_B[0]]] * acc_dims_B[inter_B[0]] + 
                                (threadIdx.x/BX + i + BK) / dim[dim_B[inter_B[1]]] * acc_dims_B[inter_B[1]];
            }
            *(sh_B + double_buffer + threadIdx.x) = *(B_start);
        }
    }
    // write C tile from register to global memory
    *C_start = reg_C;
}


// A[], (0,4,1,5), (1,3), (1, a_dims, a_dims*e_dims, a_dims*e_dims*b_dims), nd_A = 4
// B[], (5,2,4,3), (0,2), (1, f_dims, f_dims*c_dims, f_dims*c_dims*e_dims), nd_B = 4
// (a_dims,b_dims,c_dims,d_dims,e_dims,f_dims), nd = 6
// C[], (0,1) (2,3), (001111), (1, a_dims, a_dims*b_dims, a_dims*b_dims*c_dims), nd_C = 4
// acc_dims_C_A: (1,a_dims,a_dims*b_dims)   len is nd_C_A + 1
// acc_dims_C_B: (1,c_dims,c_dims*d_dims)   len is nd_C_B + 1
__global__ 
void tc_1dim_64_8X8(float A[], int dim_A[], int inter_A[], int acc_dims_A[], int nd_A, \
                    float B[], int dim_B[], int inter_B[], int acc_dims_B[], int nd_B, \
                    float C[], int dim_C_A[], int acc_dims_C_A[], int nd_C_A, int dim_C_B[], int acc_dims_C_B[], int nd_C_B, int Mask, \
                    int64_t dim[], int nd){
    // A: {dim[dim_A[0]], dim[dim_A[1]], dim[dim_A[2]], ...}
    // B: {dim[dim_B[0]], dim[dim_B[1]], dim[dim_B[2]], ...}
    // C: {dim[dim_C[0]], dim[dim_C[1]], dim[dim_C[2]], ...}
    int BY = 16, BX = 24, BK = 12;
    extern __shared__ float sh[]; 
    // shared_mem: BY*BK + BX*BK | BY*BK + BX*BK
    float *sh_A = sh;
	float *sh_B = sh + (BY*BK);  // maybe needs to modified
    
    // C (a,b  X  c,d)
    // horizontal
    //int blocks_x = (c_dims + BX - 1)/BX;
    int blocks_x = (dim[dim_C_B[0]] + BX - 1)/BX;
    
    // vertical
    //int blocks_y = (a_dims + BY - 1)/BY;
	int blocks_y = (dim[dim_C_A[0]] + BY - 1)/BY;
    
    // Load C from global memory to register file
    // Here, blockDim.x = BX * BY
    //                   + a                                               + b * a_dims                     + c * a_dims * b_dims                                               + d * a_dims * b_dims * c_dims
	//float *C_start = C + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * a_dims * b_dims + (blockIdx.x / blocks_x) * a_dims * b_dims * c_dims ;
	int idx[8]; // len is nd, which is the same with dim array
    // BY <= dim[dim_C_A[0]]
    int coeff_y = blockIdx.y % blocks_y * BY + threadIdx.x % BY; 
    // BX <= dim[dim_C_B[0]]
    int coeff_x = (blockIdx.x % blocks_x * BX + threadIdx.x / BY);
    float reg_C; 
    float *C_start;
    if (coeff_y < dim[dim_C_A[0]] && coeff_x < dim[dim_C_B[0]]){
        idx[dim_C_A[0]] = coeff_y;
        idx[dim_C_B[0]] = coeff_x;

        C_start = C + idx[dim_C_A[0]];
        for (int i=1; i<nd_C_A; ++i){
            idx[dim_C_A[i]] = blockIdx.y / (blocks_y * acc_dims_C_A[i-1]);
            C_start += idx[dim_C_A[i]] * acc_dims_C_A[i];
        }

        C_start += idx[dim_C_B[0]] * acc_dims_C_B[0] * acc_dims_C_A[nd_C_A];
        for (int i=1; i<nd_C_B; ++i){
            idx[dim_C_B[i]] = blockIdx.x / (blocks_x * acc_dims_C_B[i-1]);
            C_start += idx[dim_C_B[i]] * acc_dims_C_B[i] * acc_dims_C_A[nd_C_A];
        }
        reg_C = *C_start; 
    }    
    // A (a e b f)
    //load A from global memory to shared memory
    //                   + a                                               + b * a_dims * e_dims                     + (e) 0 * a_dims  + (f) 0 * a_dims * e_dims * b_dims
    //var/folders/6_/_434ff9j2cz1f0psvgwdf9700000gn/T/TemporaryItems/NSIRD_screencaptureui_XG5uhF/Screen\ Shot\ 2022-01-06\ at\ 2.46.30\ PM.png /float *A_start = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims + 0 * a_dims      + threadIdx.x / BY * a_dims * e_dims * b_dims;
    //int A_base = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims;   // external indices
    //float *A_start = A_base + 0 * a_dims      + (threadIdx.x / BY+0) * a_dims * e_dims * b_dims;    // internal indices of A
    float *A_start = A;
    int A_base = 0;
    if (dim_A[0]==dim_C_A[0]){
        // the first dim of A is an external dim. A1 case
        A_start += idx[dim_A[0]];
        for (int i=1; i<nd_A; ++i){
            // the dim belongs to external dims
            if ( (1<<dim_A[i]) & Mask ) A_base += idx[dim_A[i]]*acc_dims_A[i];
            // the dim belongs to internal dims
            else {
                int d = threadIdx.x / BY;
                if (d < BK){
                    // <(less than) internal dims
                    A_start += d * acc_dims_A[i];
                }
            }  
        }
    }else{
        // the first dim of A is a contraction dim. A2 case
        if (threadIdx.x / BK < BY){
            A_start += threadIdx.x % BK;
            for (int i=1; i<nd_A; ++i){
                // all the remaining dims are external dims
                A_base += idx[dim_A[i]] * acc_dims_A[i];
            }
        }
    }
    A_start += A_base;
    if ( threadIdx.x < BY*BK ){
        *(sh_A + threadIdx.x) = *(A_start);
    }
    
    // B (f c e d)
    //load B from global memory to shared memory
    //                 + c * f_dims                                               +  d * f_dims * c_dims *e_dims                      + (e) 0 * f_dims * c_dims    + (f) 0
    //float *B_start = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims + 0 * f_dims * c_dims        + threadIdx.x % BK;
    //int B_base = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims;   // external indices of B
    //float *B_start = B_base + 0 * f_dims * c_dims        + threadIdx.x % BK * 1;     // internal indices of B
    
    float *B_start = B;
    int B_base = 0;
    if (dim_B[0]!=dim_C_B[0]){
        // the first dim of B is a contraction dim. B1 case
        // in this case, we have select it as the first iteration contraction dim previously
        if (threadIdx.x / BK < BX){
            B_start += threadIdx.x % BK;
            for (int i=1; i<nd_B; ++i){
                // all the remaining dims are external dims
                B_base += idx[dim_B[i]]*acc_dims_B[i];
            }
        }
    }else{
        // the first dim of B is an external dim. B2 case
        B_start += idx[dim_B[0]];
        for (int i=1; i<nd_B; ++i){
            // the dim belongs to external dims
            if ( (1<<dim_B[i]) & Mask ) B_base += idx[dim_B[i]]*acc_dims_B[i];
            else{
                // the dim belongs to internal dims
                int d = threadIdx.x / BX;
                if (d < BK){
                    B_start += d * acc_dims_B[i];
                }
            } 
        }
    }
    
    B_start += B_base;
    if (threadIdx.x < BX*BK){
        *(sh_B + threadIdx.x) = *(B_start);
    }
    
    // compute the number of contraction dims
    int prod = dim[dim_A[inter_A[0]]];
    
    // shared_mem: (BY*BK + BX*BK) * 2
    //        0    ~ (BY-1)*BK: A
    // (BY-1)*BK+1 ~ (BY-1)*BK+(BX-1)*BK: B
    int double_buffer = 0;
    float reg_A,reg_B;
    
    int A_offset;
	int B_offset;
    for (int i=0; i<prod; i += BK){
        __syncthreads();
        if (dim_C_A[0]==dim_A[0] && dim_B[0]==inter_B[0]){
            A_offset = double_buffer + threadIdx.x%BY;
		    B_offset = double_buffer + (threadIdx.x/BY) * BK;
        }else if (dim_C_A[0]==dim_A[0]){
            A_offset = double_buffer + threadIdx.x%BY;
		    B_offset = double_buffer + threadIdx.x/BY;
        }else if (dim_B[0]==inter_B[0]){
            A_offset = double_buffer + (threadIdx.x%BY) * BK;
		    B_offset = double_buffer + (threadIdx.x/BY) * BK;
        }else{
            A_offset = double_buffer + (threadIdx.x%BY) * BK;
		    B_offset = double_buffer + threadIdx.x/BY;
        }
        for (int k=0; k<BK; ++k){
            // read A tile from shared memory to registers
            reg_A = sh_A[A_offset];
            // read B tile from shared memory to registers
            reg_B = sh_B[B_offset];
            // perform FMA
            reg_C = fma(reg_A, reg_B, reg_C);

            A_offset += BY;
			B_offset += 1;
        }
        // BX*BK+BY*BK is power of 2
        double_buffer ^= (BX*BK+BY*BK);
        if (i+BK < prod){
            if (dim_C_A[0]==dim_A[0]){
                // external indices
                A_start = A + A_base + (threadIdx.x/BY + i + BK) % prod * acc_dims_A[inter_A[0]];
            }else{
                // internal indices of A. A2 case
                A_start = A + A_base + (threadIdx.x%BK + i + BK) % prod * acc_dims_A[inter_A[0]]; 
            }
            if ( threadIdx.x < BY*BK ){
                *(sh_A + double_buffer + threadIdx.x) = *(A_start);
            }
            if (dim_B[0]==inter_B[0]){
                // internal indices of B. B1 case
                B_start = B + B_base + (threadIdx.x%BK + i + BK) % prod * acc_dims_B[0];  
            }else{
                // external indices
                B_start = B + B_base + (threadIdx.x/BX + i + BK) % prod * acc_dims_B[inter_B[0]];
            }
            if (threadIdx.x < BX*BK){
                *(sh_B + double_buffer + threadIdx.x) = *(B_start);
            }
        }
    }
    // write C tile from register to global memory
    *C_start = reg_C;
    
}
void run_cuTC(Tensor &AA, Tensor &BB, Tensor &CC){
    std::vector<int> modeC(CC.get_Mode());
    std::vector<int> modeA(AA.get_Mode());
    std::vector<int> modeB(BB.get_Mode());
    for (auto &it:modeC){
        it = it-'a';
    } 
    for (auto &it:modeB){
        it = it-'a';
    } 
    for (auto &it:modeA){
        it = it-'a';
    } 
    std::vector<int64_t> extentC(CC.get_Ext());
    //for (auto mode : modeC)
    //    extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA(AA.get_Ext());
    //for (auto mode : modeA)
    //    extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB(BB.get_Ext());
    //for (auto mode : modeB)
    //    extentB.push_back(extent[mode]);

    std::unordered_map<int, int64_t> extent;
    for (int i=0; i<modeC.size(); ++i){
        if (!extent.count(modeC[i])){
            extent[modeC[i]] = extentC[i]; 
        }else if(extent[modeC[i]]!=extentC[i]){
            std::cout<<"C extent error!"<<std::endl;
            exit(1);
        }
    }
    for (int i=0; i<modeA.size(); ++i){
        if (!extent.count(modeA[i])){
            extent[modeA[i]] = extentA[i]; 
        }else if(extent[modeA[i]]!=extentA[i]){
            std::cout<<"A extent error!"<<std::endl;
            exit(1);
        }
    }
    for (int i=0; i<modeB.size(); ++i){
        if (!extent.count(modeB[i])){
            extent[modeB[i]] = extentB[i]; 
        }else if(extent[modeB[i]]!=extentB[i]){
            std::cout<<"B extent error!"<<std::endl;
            exit(1);
        }
    }
    int64_t *dim = (int64_t *)malloc(extent.size()*sizeof(int64_t));
    for (auto it:extent){
        dim[it.first] = it.second;
    }
   
    size_t elementsA = 1;
    for (auto num : extentA)
        elementsA *= num;
    size_t elementsB = 1;
    for (auto num : extentB)
        elementsB *= num;
    size_t elementsC = 1;
    for (auto num : extentC)
        elementsC *= num;

    float *A = (float *) malloc(sizeof(float) * elementsA);
    float *B = (float *) malloc(sizeof(float) * elementsB);
    float *C = (float *) malloc(sizeof(float) * elementsC);

    //if (A == NULL || B == NULL || C == NULL)
    //{
    //    printf("Error: Host allocation of A or C.\n");
    //    return -1;
    //}

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = AA.data[i];
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = BB.data[i];
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = CC.data[i];

    int32_t *dim_C = (int32_t *)malloc(modeC.size()*sizeof(int32_t));
    for (int i=0; i<modeC.size(); ++i){
        dim_C[i] = modeC[i];
    }
    tensorContraction_host(A, modeA.data(), modeA.size(),
                           B, modeB.data(), modeB.size(),
                           C, dim_C, modeC.size(),
                           dim, extent.size());
}
void tensorContraction_host(float A[], int dim_A[], int nd_A, \
                    float B[], int dim_B[], int nd_B, \
                    float C[], int32_t dim_C[], int nd_C, \
                    int64_t dim[], int nd){
    // A: {dim[dim_A[0]], dim[dim_A[1]], dim[dim_A[2]], ...}
    // B: {dim[dim_B[0]], dim[dim_B[1]], dim[dim_B[2]], ...}
    // C: {dim[dim_C[0]], dim[dim_C[1]], dim[dim_C[2]], ...}
    int kernel_choice = (nd_A + nd_B - nd_C)/2;
    int mask = 0;
    int size_C = 1;
    for (int i=0; i<nd_C; ++i){
        mask |= (1<<dim_C[i]);
        size_C *= dim_C[i];
    }
    int bit_A = 0;
    int size_A = 1;
    for (int i = 0; i<nd_A; i++){
        bit_A |= (1<<dim_A[i]);
        size_A *= dim_A[i];
    }
    int bit_B = 0;
    int size_B = 1;
    for (int i = 0; i<nd_B; i++){
        bit_B |= (1<<dim_B[i]);
        size_B *= dim_B[i];
    }
    
    int *acc_dims_A_host = (int *) malloc(nd_A * sizeof(int));
    acc_dims_A_host[0] = 1;
    int *inter_A_host = (int *) malloc(kernel_choice * sizeof(int));;
    int k = 0;
    if (!( (mask & bit_A) & (1<<dim_A[0]) )) inter_A_host[k++] = 0;  
    for (int i = 1; i<nd_A; i++){
        if ( !( (mask & bit_A) & (1<<dim_A[i]) ) ) inter_A_host[k++] = i; 
        acc_dims_A_host[i] = acc_dims_A_host[i-1] * dim[dim_A[i-1]];
    }
    
    int *acc_dims_B_host = (int *) malloc(nd_B * sizeof(int));
    acc_dims_B_host[0] = 1;
    int *inter_B_host = (int *) malloc(kernel_choice * sizeof(int));;
    k = 0;
    if (!( (mask & bit_B) & (1<<dim_B[0]) )) inter_B_host[k++] = 0;  
    for (int i = 1; i<nd_B; i++){
        if ( !( (mask & bit_B) & (1<<dim_B[i]) ) ) inter_B_host[k++] = i; 
        acc_dims_B_host[i] = acc_dims_B_host[i-1] * dim[dim_B[i-1]];
    }
    int nd_C_A = nd_A-kernel_choice;
    int nd_C_B = nd_B-kernel_choice;
    int *acc_dims_C_A_host = (int *) malloc((nd_C_A+1) * sizeof(int));
    int *acc_dims_C_B_host = (int *) malloc((nd_C_B+1) * sizeof(int));
    acc_dims_C_A_host[0] = 1;
    acc_dims_C_B_host[0] = 1;
    int *dim_C_A_host = (int *) malloc(nd_C_A * sizeof(int));
    int *dim_C_B_host = (int *) malloc(nd_C_B * sizeof(int));
    int k1 = 0, k2 = 0;
    
    for (int i = 0; i<nd_A; i++){
        if ( mask & (1<<dim_A[i]) ){
            dim_C_A_host[k1++] = dim_A[i];
        }
    }
    for (int i = 1; i<=nd_C_A; i++){
        acc_dims_C_A_host[i] = acc_dims_C_A_host[i-1] * dim[dim_C_A_host[i-1]];
    }
    
    for (int i = 0; i<nd_B; i++){
        if ( mask & (1<<dim_B[i]) ){
            dim_C_B_host[k2++] = dim_B[i];
        }
    }
    for (int i = 1; i<=nd_C_B; i++){
        acc_dims_C_B_host[i] = acc_dims_C_B_host[i-1] * dim[dim_C_B_host[i-1]];
    }
    
    dim3 grid_size(0,0,1);
    int BX = 24;
    int blocks_x = (dim[dim_C_B_host[0]] + BX - 1) / BX;
    int prod = 1;
    for (int i=1; i<nd_C_B; ++i){
        prod *= dim[dim_C_B_host[i]];
    }
    grid_size.x = prod * blocks_x;
    int BY = 16;
    int blocks_y = (dim[dim_C_A_host[0]] + BY - 1) / BY;
    prod = 1;
    for (int i=1; i<nd_C_A; ++i){
        prod *= dim[dim_C_A_host[i]];
    }
    grid_size.y = prod * blocks_y;

    int BK = 12;
    dim3 block_size(BX*BY,1,1);
    int shared_mem_size = sizeof(float)*2*(BX*BK+BY*BK);
    
    // transfer data to the device 
    // tensor A
    float *A_device;
    ErrChk(hipMalloc((void**)&A_device, size_A*sizeof(float)));
    ErrChk(hipMemcpy(A_device, A, size_A*sizeof(float), hipMemcpyHostToDevice));
    
    int *dim_A_device;
    ErrChk(hipMalloc((void**)&dim_A_device, nd_A*sizeof(int)));
    ErrChk(hipMemcpy(dim_A_device, dim_A, nd_A*sizeof(int), hipMemcpyHostToDevice));

    int *inter_A_device;
    ErrChk(hipMalloc((void**)&inter_A_device, kernel_choice*sizeof(int)));
    ErrChk(hipMemcpy(inter_A_device, inter_A_host, kernel_choice*sizeof(int), hipMemcpyHostToDevice));

    int *acc_dims_A_device;
    ErrChk(hipMalloc((void**)&acc_dims_A_device, nd_A*sizeof(int)));
    ErrChk(hipMemcpy(acc_dims_A_device, acc_dims_A_host, nd_A*sizeof(int), hipMemcpyHostToDevice));
    
    // tensor B
    float *B_device;
    ErrChk(hipMalloc((void**)&B_device, size_B*sizeof(float)));
    ErrChk(hipMemcpy(B_device, B, size_B*sizeof(float), hipMemcpyHostToDevice));

    int *dim_B_device;
    ErrChk(hipMalloc((void**)&dim_B_device, nd_B*sizeof(int)));
    ErrChk(hipMemcpy(dim_B_device, dim_B, nd_B*sizeof(int), hipMemcpyHostToDevice));

    int *inter_B_device;
    ErrChk(hipMalloc((void**)&inter_B_device, kernel_choice*sizeof(int)));
    ErrChk(hipMemcpy(inter_B_device, inter_B_host, kernel_choice*sizeof(int), hipMemcpyHostToDevice));

    int *acc_dims_B_device;
    ErrChk(hipMalloc((void**)&acc_dims_B_device, nd_B*sizeof(int)));
    ErrChk(hipMemcpy(acc_dims_B_device, acc_dims_B_host, nd_B*sizeof(int), hipMemcpyHostToDevice));


    // tensor C
    float *C_device;
    ErrChk(hipMalloc((void**)&C_device, size_C*sizeof(float)));
    ErrChk(hipMemcpy(C_device, C, size_C*sizeof(float), hipMemcpyHostToDevice));

    int *dim_C_A_device;
    ErrChk(hipMalloc((void**)&dim_C_A_device, nd_C_A*sizeof(int)));
    ErrChk(hipMemcpy(dim_C_A_device, dim_C_A_host, nd_C_A*sizeof(int), hipMemcpyHostToDevice));

    int *acc_dims_C_A_device;
    ErrChk(hipMalloc((void**)&acc_dims_C_A_device, (nd_C_A+1)*sizeof(int)));
    ErrChk(hipMemcpy(acc_dims_C_A_device, acc_dims_C_A_host, (nd_C_A+1)*sizeof(int), hipMemcpyHostToDevice));

    int *dim_C_B_device;
    ErrChk(hipMalloc((void**)&dim_C_B_device, nd_C_B*sizeof(int)));
    ErrChk(hipMemcpy(dim_C_B_device, dim_C_B_host, nd_C_B*sizeof(int), hipMemcpyHostToDevice));

    int *acc_dims_C_B_device;
    ErrChk(hipMalloc((void**)&acc_dims_C_B_device, (nd_C_B+1)*sizeof(int)));
    ErrChk(hipMemcpy(acc_dims_C_B_device, acc_dims_C_B_host, (nd_C_B+1)*sizeof(int), hipMemcpyHostToDevice));

    // dim
    int64_t *dim_device;
    ErrChk(hipMalloc((void**)&dim_device, nd*sizeof(int64_t)));
    ErrChk(hipMemcpy(dim_device, dim, nd*sizeof(int64_t), hipMemcpyHostToDevice));

    // call the kernel
    if (kernel_choice==1){
        // the contraction-dim is 1
        tc_1dim_64_8X8<<<grid_size,block_size,shared_mem_size>>>(A_device, dim_A_device, inter_A_device, acc_dims_A_device, nd_A, \
                                                                 B_device, dim_B_device, inter_B_device, acc_dims_B_device, nd_B, \
                                                                 C_device, dim_C_A_device, acc_dims_C_A_device, nd_C_A, dim_C_B_device, acc_dims_C_B_device, nd_C_B, mask, \
                                                                 dim_device, nd);
    }else{
        // the contraction-dim is 2
        tc_2dims_64_8X8<<<grid_size,block_size,shared_mem_size>>>(A_device, dim_A_device, inter_A_device, acc_dims_A_device, nd_A, \
                                                                  B_device, dim_B_device, inter_B_device, acc_dims_B_device, nd_B, \
                                                                  C_device, dim_C_A_device, acc_dims_C_A_device, nd_C_A, dim_C_B_device, acc_dims_C_B_device, nd_C_B, mask, \
                                                                  dim_device, nd);
    }
    /*
    // tensor C_final
    float *C_final_device;
    ErrChk(hipMalloc((void**)&C_final_device, size_C*sizeof(float)));

    // permutation
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;
    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.f;

    hiptensorHandle_t handle;
    HANDLE_ERROR(cutensorInit(&handle));
    hiptensorTensorDescriptor_t descC_final;
    int64_t *extent_C_final = (int64_t *) malloc(nd_C * sizeof(int64_t));
    for (int i = 0; i<nd_C; i++){
        extent_C_final[i] = dim[dim_C[i]];
    }
    HANDLE_ERROR(hiptensorInitTensorDescriptor( &handle,
                 &descC_final,
                 nd_C,
                 extent_C_final,
                 NULL,
                 typeC, HIPTENSOR_OP_IDENTITY));
    
    int64_t *extent_C_device = (int64_t *) malloc(nd_C * sizeof(int64_t));
    int32_t *dim_C_device = (int32_t *) malloc(nd_C * sizeof(int32_t));
    int k_C = 0;
    for (int i = 0; i<nd_C_A; i++){
        extent_C_device[k_C] = dim[dim_C_A_host[i]];
        dim_C_device[k_C++] = dim_C_A_host[i];
    }
    
    for (int i = 0; i<nd_C_B; i++){
        extent_C_device[k_C] = dim[dim_C_B_host[i]];
        dim_C_device[k_C++] = dim_C_B_host[i];
    }
    hiptensorTensorDescriptor_t descC_device;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( &handle,
                 &descC_device,
                 nd_C,
                 extent_C_device,
                 NULL,
                 typeC, HIPTENSOR_OP_IDENTITY));
    hiptensorPermutation(&handle, (void*) &alpha, 
                        C_device, &descC_device, dim_C_device, 
                        C_final_device, &descC_final, dim_C, typeC, 0);

    ErrChk(hipMemcpy(C, C_final_device, size_C*sizeof(float), hipMemcpyDeviceToHost));
   */ 
    // varify the correctness
    // in previous level function

    free(acc_dims_A_host);
	free(inter_A_host);
	free(acc_dims_B_host);
	free(inter_B_host);
    
    free(acc_dims_C_A_host);
    free(acc_dims_C_B_host);
	free(dim_C_A_host);
    free(dim_C_B_host);

    /*
    ErrChk(hipFree(A_device));
    ErrChk(hipFree(dim_A_device));
    ErrChk(hipFree(inter_A_device));
    ErrChk(hipFree(acc_dims_A_device));

    ErrChk(hipFree(B_device));
    ErrChk(hipFree(dim_B_device));
    ErrChk(hipFree(inter_B_device));
    ErrChk(hipFree(acc_dims_B_device));

    ErrChk(hipFree(C_device));
    ErrChk(hipFree(dim_C_A_device));
    ErrChk(hipFree(acc_dims_C_A_device));
    ErrChk(hipFree(dim_C_B_device));
    ErrChk(hipFree(acc_dims_C_B_device));
*/
}
/*
__global__ 
void tensor_contraction_2contraction_Dim_64_8x8(float* A[], int dim_A[], int nd_A, \
                    float* B[], int dim_B[], int nd_B, \
                    float* C[], int dim_C[], int nd_C, \
                    int dim[], int nd, \
                    int dim_intra[], int nd_intra){
    // A: {dim[dim_A[0]], dim[dim_A[1]], dim[dim_A[2]], ...}
    // B: {dim[dim_B[0]], dim[dim_B[1]], dim[dim_B[2]], ...}
    // C: {dim[dim_C[0]], dim[dim_C[1]], dim[dim_C[2]], ...}
    int BY = 8, BX = 8, BK = 8;
    extern __shared__ float sh[]; 
    // shared_mem: BY*BK + BX*BK | BY*BK + BX*BK
    float *sh_A = sh;
	float *sh_B = sh + (BY*BK);  // maybe needs to modified
    
    // C (a,b  X  c,d)
    // horizontal
    int blocks_x = (c_dims + BX - 1)/BX;
    //int block_base_x = blockIDx.x / blocks_x * c_dims + blockIdx.x % blocks_x * BX; // <= N
    
    // vertical
    int blocks_y = (a_dims + BY - 1)/BY;
	//int block_base_y = blocIDx.y / blocks_y * a_dims + blockIdx.y % blocks_y * BY; // <= M                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                lockIdx.x * BY;
    
    // Load C from global memory to register file
    // Here, blockDim.x = BX * BY
    //                 + a                                               + b * a_dims                     + c * a_dims * b_dims                                               + d * a_dims * b_dims * c_dims
	float *C_start = C + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * a_dims * b_dims + (blockIdx.x / blocks_x) * a_dims * b_dims * c_dims ;
	reg_C = *C_start; 
    
    // A (a e b f)
    //load A from global memory to shared memory
    //                   + a                                               + b * a_dims * e_dims                     + (e) 0 * a_dims  + (f) 0 * a_dims * e_dims * b_dims
    //float *A_start = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims + 0 * a_dims      + threadIdx.x / BY * a_dims * e_dims * b_dims;
    int A_base = A + (blockIdx.y % blocks_y * BY + threadIdx.x % BY) + blockIdx.y / blocks_y * a_dims * e_dims;   // external indices
    float *A_start = A_base + 0 * a_dims      + (threadIdx.x / BY+0) * a_dims * e_dims * b_dims;    // internal indices of A
    *(sh_A + threadIdx.x) = *(A_start);
    // B (f c e d)
    //load B from global memory to shared memory
    //                 + c * f_dims                                               +  d * f_dims * c_dims *e_dims                      + (e) 0 * f_dims * c_dims    + (f) 0
    //float *B_start = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims + 0 * f_dims * c_dims        + threadIdx.x % BK;
    int B_base = B + (blockIdx.x % blocks_x * BX + threadIdx.x / BY) * f_dims + (blockIdx.x / blocks_x) * f_dims * c_dims *e_dims;   // external indices of B
    float *B_start = B_base + 0 * f_dims * c_dims        + threadIdx.x % BK * 1;     // internal indices of B
    *(sh_B + threadIdx.x) = *(B_start);
    // shared_mem: (BY*BK + BX*BK) * 2
    //        0    ~ (BY-1)*BK: A
    // (BY-1)*BK+1 ~ (BY-1)*BK+(BX-1)*BK: B
    int double_buffer = 0; 
    for (int i=0; i<e_dims * f_dims; i += BK){
        int A_offset = double_buffer + (threadIdx.x%BY);
		int B_offset = double_buffer + (threadIdx.x/BY)*BK;
            
        for (int k=0; k<BK; ++k){
            // read A tile from shared memory to registers
            reg_A = sh_A[A_offset];
            // read B tile from shared memory to registers
            reg_B = sh_B[B_offset];
            // perform FMA
            reg_C = fma(reg_A, reg_B, reg_C);

            A_offset += BY;
			B_offset += 1;
        }
        // BX*BK+BY*BK = 8*8+8*8 = 128
        double_buffer ^= (BX*BK+BY*BK);
        if (i+BK < e_dims * f_dims){
            // internal indices of A
            f_A = (threadIdx.x/BY + i + BK) % f_dims;
            e_A = (threadIdx.x/BY + i + BK) / f_dims;
            //               + e * a_dims + f * a_dims * e_dims * b_dims
            A_start = A_base + e * a_dims + f * a_dims * e_dims * b_dims; 
            *(sh_A + double_buffer + threadIdx.x) = *(A_start);

            // internal indices of B 
            f_B = (threadIdx.x%BK + i + BK) % f_dims;
            e_B = (threadIdx.x%BK + i + BK) / f_dims; 
            //               + e * f_dims * c_dims + f
            B_start = B_base + e * f_dims * c_dims + f; 
            *(sh_B + double_buffer + threadIdx.x) = *(B_start);
        }
    }
    // write C tile from register to global memory
    *C_start = reg_C;
}
*/
